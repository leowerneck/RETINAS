#include "image_analysis.h"

__host__ static inline
void complex_matrix_multiply(
    hipblasHandle_t h,
    const int m,
    const int p,
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  Performs the matrix multiplication C = A.B. This function
   *  uses cuBLAS.
   *
   *  Inputs
   *  ------
   *    h : cuBLAS handle object.
   *    m : Common dimension of matrices A and C.
   *    p : Common dimension of matrices A and B.
   *    n : Common dimension of matrices B and C.
   *    A : Flattened matrix of dimension m x p.
   *    B : Flattened matrix of dimension p x n.
   *    C : Flattened matrix of dimension m x n. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const COMPLEX alpha = MAKE_COMPLEX(1.0, 0.0);
  const COMPLEX beta  = MAKE_COMPLEX(0.0, 0.0);
  CUBLASCGEMM(h, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, p,
              &alpha, A, m, B, p, &beta, C, m);
}

__host__ static inline
void complex_matrix_multiply_tt(
    hipblasHandle_t h,
    const int m,
    const int p,
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  Performs the matrix multiplication C = (A.B)^T = BT.AT, where
   *  AT/BT is the transpose of A/B. This function uses cuBLAS.
   *
   *  Inputs
   *  ------
   *    h : cuBLAS handle object.
   *    m : Common dimension of matrices A and C.
   *    p : Common dimension of matrices A and B.
   *    n : Common dimension of matrices B and C.
   *    A : Flattened matrix of dimension m x p. AT is p x m.
   *    B : Flattened matrix of dimension p x n. BT is n x p.
   *    C : Flattened matrix of dimension n x m. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const COMPLEX alpha = MAKE_COMPLEX(1.0,0.0);
  const COMPLEX beta  = MAKE_COMPLEX(0.0,0.0);
  CUBLASCGEMM(h, HIPBLAS_OP_T, HIPBLAS_OP_T, n, m, p,
              &alpha, B, p, A, m, &beta, C, m);
}

extern "C" __host__
void upsample_around_displacements(
    state_struct *restrict state,
    REAL *restrict displacements ) {
  /*
   *  Upsample the region around displacements and recompute
   *  them with subpixel precision.
   *
   *  Inputs
   *  ------
   *    state         : The CUDA state object containing all required data.
   *    displacements : Array of horizontal and vertical displacements. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  // Note: At the beginning of this function, aux_array1 stores the
  //       image product, while aux_array2 and aux_array3 are empty.
  //
  // Step 1: Set basic variables
  const int Nh = state->N_horizontal;
  const int Nv = state->N_vertical;
  const REAL upsample_factor = state->upsample_factor;

  // Step 2: Adjust the displacements based on the upsample factor
  for(int i=0;i<2;i++) displacements[i] = ROUND(displacements[i] * upsample_factor)/upsample_factor;

  // Step 3: Set size of upsampled region
  const REAL upsampled_region_size = CEIL(upsample_factor * 1.5);

  // Step 4: Center of the output array at dftshift+1
  const REAL dftshift = round_towards_zero(upsampled_region_size / 2.0);

  // Step 5: Compute upsample region offset
  REAL sample_region_offset[2];
  for(int i=0;i<2;i++) sample_region_offset[i] = dftshift - displacements[i]*upsample_factor;

  // Step 6: Upsampled size
  const int S  = (int)upsampled_region_size;

  // Step 7: Compute the horizontal kernel
  compute_horizontal_kernel(sample_region_offset, state);

  // Step 8: Contract the horizontal kernel with the conjugate of the image product
  complex_conjugate_2d(Nh, Nv, state->aux_array1);
  // Note: aux_array1 contains the complex conjugate of the image product,
  //       aux_array2 contains the horizontal kernel, and
  //       aux_array3 will contain the matrix product of aux_array2 and aux_array1.
  complex_matrix_multiply(state->cublasHandle,S,Nh,Nv,state->aux_array2,state->aux_array1,state->aux_array3);

  // Step 9: Compute the vertical kernel
  compute_vertical_kernel(sample_region_offset, state);

  // Step 10: Now contract the result of Step 8 with the vertical kernel to get the upsampled image
  // Note: aux_array1 will contains the upsampled image,
  //       aux_array2 contains the vertical kernel, and
  //       aux_array3 is the same as in Step 8.
  complex_matrix_multiply_tt(state->cublasHandle,S,Nv,S,state->aux_array3,state->aux_array2,state->aux_array1);
}
