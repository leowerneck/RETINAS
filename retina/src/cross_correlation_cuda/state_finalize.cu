#include "image_analysis.h"

extern "C" __host__
void state_finalize( state_struct *restrict state ) {
  /*
   *  Free memory for all pointers in the C state object,
   *  as well as the memory allocated for the object.
   *
   *  Inputs
   *  ------
   *    state : Pointer to the state object.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  // Step 1: Free memory for all host arrays
  free(state->host_aux_array);

  // Step 2: Free memory for all device arrays
  hipFree(state->aux_array_int);
  hipFree(state->aux_array_real);
  hipFree(state->aux_array1);
  hipFree(state->aux_array2);
  hipFree(state->aux_array3);
  hipFree(state->new_image_time);
  hipFree(state->new_image_freq);
  hipFree(state->new_image_time_squared);
  hipFree(state->eigenframe_freq);

  // Step 3: Destroy FFT plans
  FFT_DESTROY_PLAN(state->fft2_plan);

  // Step 4: Destroy cuBLAS handle
  hipblasDestroy(state->cublasHandle);

  // Step 5: Free memory allocated for the state struct
  free(state);

  info("Successfully finalized state object\n");
}