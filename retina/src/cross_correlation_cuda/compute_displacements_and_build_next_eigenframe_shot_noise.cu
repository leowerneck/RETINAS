#include "image_analysis.h"

extern "C" __host__
void compute_displacements_and_build_next_eigenframe_shot_noise(
    state_struct *restrict state,
    REAL *restrict displacements ) {
  /*
   *  Obtain the displacement between the new and reference images.
   *
   *  Inputs
   *  ------
   *    state         : The CUDA state object, containing the new and reference images.
   *    displacements : Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  // Step 1: Compute the displacements via cross-correlation
  cross_correlate_ref_and_new_images(state);

  // const int Nh = state->N_horizontal;
  // const int Nv = state->N_vertical;
  // printf("I^2:\n");
  // print_2d_array_complex(Nh, Nv, state->new_image_time);
  // printf("F[I^2]:\n");
  // print_2d_array_complex(Nh, Nv, state->new_image_freq);
  // printf("F[1/E]:\n");
  // print_2d_array_complex(Nh, Nv, state->eigenframe_freq);
  // printf("Image product (aux_array1):\n");
  // print_2d_array_complex(Nh, Nv, state->aux_array1);
  // printf("Cross-correlation (aux_array2):\n");
  // print_2d_array_complex(Nh, Nv, state->aux_array2);

  // Step 2: Get the full pixel estimate of the displacements
  displacements_full_pixel_estimate_shot_noise(state, displacements);

  // printf("Full pixel displacements: %.15e %.15e\n", displacements[0], displacements[1]);

  // Step 3: Compute the displacements using upsampling
  if( (int)(state->upsample_factor+0.5) > 1 ) {
    upsample_around_displacements(state, displacements);
    displacements_sub_pixel_estimate_shot_noise(state, displacements);
  }

  // Step 4: Before building the next eigenframe we must compute
  //         the FFT of the reciprocal of the new image
  FFT_EXECUTE_DFT(state->fft2_plan,
                  state->reciprocal_new_image_time,
                  state->new_image_freq,
                  HIPFFT_FORWARD);

  // const int Nh = state->N_horizontal;
  // const int Nv = state->N_vertical;
  // printf("(1/I+s):\n");
  // print_2d_array_complex(Nh, Nv, state->reciprocal_new_image_time);
  // printf("F[(1/I+s)]:\n");
  // print_2d_array_complex(Nh, Nv, state->new_image_freq);

  // Step 5: Build next eigenframe
  build_next_eigenframe(displacements, state);
}
