#include "image_analysis.h"

static inline
int round_towards_zero(const REAL x) {
  /*
   *  Rounds a number to the nearest integer towards zero.
   *
   *  Inputs
   *  ------
   *    x : Number to be rounded.
   *
   *  Returns
   *  -------
   *    y : Number rounded towards zero.
   */
  if( x > 0.0 )
    return FLOOR(x);
  else
    return CEIL(x);
}

__host__ static inline
void complex_matrix_multiply(
    hipblasHandle_t cublasHandle,
    const int m,
    const int p,
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  Performs the matrix multiplication C = A.B. This function
   *  uses cuBLAS.
   *
   *  Inputs
   *  ------
   *    m : Common dimension of matrices A and C.
   *    p : Common dimension of matrices A and B.
   *    n : Common dimension of matrices B and C.
   *    A : Flattened matrix of dimension m x p.
   *    B : Flattened matrix of dimension p x n.
   *    C : Flattened matrix of dimension m x n. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const COMPLEX alpha = MAKE_COMPLEX(1.0,0.0);
  const COMPLEX beta  = MAKE_COMPLEX(0.0,0.0);
  CUBLASCGEMM(cublasHandle,HIPBLAS_OP_N,HIPBLAS_OP_N,
              m,n,p,&alpha,A,m,B,p,&beta,C,m);
}

__host__ static inline
void complex_matrix_multiply_tt(
    hipblasHandle_t cublasHandle,
    const int m,
    const int p,
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  Performs the matrix multiplication C = (A.B)^T = BT.AT, where
   *  AT/BT is the transpose of A/B. This function uses cuBLAS.
   *
   *  Inputs
   *  ------
   *    m : Common dimension of matrices A and C.
   *    p : Common dimension of matrices A and B.
   *    n : Common dimension of matrices B and C.
   *    A : Flattened matrix of dimension m x p. AT is p x m.
   *    B : Flattened matrix of dimension p x n. BT is n x p.
   *    C : Flattened matrix of dimension n x m. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const COMPLEX alpha = MAKE_COMPLEX(1.0,0.0);
  const COMPLEX beta  = MAKE_COMPLEX(0.0,0.0);
  CUBLASCGEMM(cublasHandle,HIPBLAS_OP_T,HIPBLAS_OP_T,
              n,m,p,&alpha,B,p,A,m,&beta,C,m);
}

extern "C" __host__
void upsample_and_compute_subpixel_displacements(
    state_struct *restrict state,
    REAL *restrict displacements ) {
  /*
   *  Upsample the region around displacements and recompute
   *  them with subpixel precision.
   *
   *  Inputs
   *  ------
   *    state         : The CUDA state object containing all required data.
   *    displacements : Array of horizontal and vertical displacements. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  // Note: At the beginning of this function, aux_array1 stores the
  //       image product, while aux_array2 and aux_array3 are empty.
  //
  // Step 1: Set basic variables
  const int Nh   = state->N_horizontal;
  const int Nv   = state->N_vertical;
  const int NhNv = Nh*Nv;
  const REAL upsample_factor = state->upsample_factor;

  // Step 2: Adjust the displacement based on the upsample factor
  for(int i=0;i<2;i++) displacement[i] = ROUND(displacement[i] * upsample_factor)/upsample_factor;

  // Step 3: Set size of upsampled region
  const REAL upsampled_region_size = CEIL(upsample_factor * 1.5);

  // Step 4: Center of the output array at dftshift+1
  const REAL dftshift = round_towards_zero(upsampled_region_size / 2.0);

  // Step 5: Compute upsample region offset
  REAL sample_region_offset[2];
  for(int i=0;i<2;i++) sample_region_offset[i] = dftshift - displacement[i]*upsample_factor;

  // Step 6: Upsampled size
  const int S  = (int)upsampled_region_size;
  const int SS = S*S;

  // Step 7: Compute the horizontal kernel
  compute_horizontal_kernel(sample_region_offset, state);

  // Step 8: Contract the horizontal kernel with the conjugate of the image product
  complex_conjugate_2d(Nh, Nv, state->aux_array1);
  // Note: aux_array1 contains the complex conjugate of the image product,
  //       aux_array2 contains the horizontal kernel, and
  //       aux_array3 will contain the matrix product of aux_array2 and aux_array1.
  complex_matrix_multiply(state->cublasHandle,S,Nh,Nv,state->aux_array2,state->aux_array1,state->aux_array3);

  // Step 9: Compute the vertical kernel
  compute_vertical_kernel(sample_region_offset, state);

  // Step 10: Now contract the result of Step 8 with the vertical kernel to get the upsampled image
  // Note: aux_array1 will contains the upsampled image,
  //       aux_array2 contains the vertical kernel, and
  //       aux_array3 is the same as in Step 8.
  complex_matrix_multiply_tt(state->cublasHandle,S,Nv,S,state->aux_array3,state->aux_array2,state->aux_array1);

  // Step 10: Get maximum of upsampled image
  absolute_value_2d(S, S, state->aux_array1, state->aux_array_real);
  const int idx_max = find_maxima(state->cublasHandle,SS,state->aux_array_real);
  const int i_max   = idx_max/S;
  const int j_max   = idx_max - i_max*S;

  // Step 11: Update the displacement
  displacement[0] += ((REAL)i_max - dftshift)/upsample_factor;
  displacement[1] += ((REAL)j_max - dftshift)/upsample_factor;
}
