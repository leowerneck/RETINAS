#include "image_analysis.h"

extern "C" __host__
void state_finalize( state_struct *restrict state ) {
  /*
   *  Free memory for all pointers in the C state object,
   *  as well as the memory allocated for the object.
   *
   *  Inputs
   *  ------
   *    state : Pointer to the state object.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  // Step 1: Free memory for all device arrays
  hipFree(state->aux_array_int);
  hipFree(state->aux_array_real);
  hipFree(state->aux_array1);
  hipFree(state->aux_array2);
  hipFree(state->aux_array3);
  hipFree(state->new_image_time);
  hipFree(state->new_image_freq);
  hipFree(state->reciprocal_new_image_time);
  hipFree(state->ref_image_freq);
  hipFree(state->image_sum_freq);

  // Step 2: Destroy FFT plans
  FFT_DESTROY_PLAN(state->fft2_plan);

  // Step 3: Destroy cuBLAS handle
  hipblasDestroy(state->cublasHandle);

  // Step 4: Free memory allocated for the state struct
  free(state);

  info("Successfully finalized state object\n");
}