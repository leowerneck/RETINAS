void test_if_gpu_works_() {
  printf("success!\n");
}

bool test_if_gpu_works() {
  printf("Testing if GPU works... ");
  fflush(stdout);
  test_if_gpu_works_<<<1, 1>>>();
  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("failed!\n");
    printf("Error name: %s", hipGetErrorName(err));
    printf("Error msg : %s", hipGetErrorString(err));
    return false;
  }

  return true;
}
