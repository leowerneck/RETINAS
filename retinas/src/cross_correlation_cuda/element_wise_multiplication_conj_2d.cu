#include "hip/hip_runtime.h"
#include "image_analysis.h"

__global__
static void element_wise_multiplication_conj_1d_gpu(
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  Computes the element-wise multiplication C = A.B^{*},
   *  where B^{*} is the complex conjugate of B.
   *
   *  Inputs
   *  ------
   *    n : Size of the arrays.
   *    A : Complex array of size n.
   *    B : Complex array of size n.
   *    C : Complex array of size n. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const int index  = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for(int i=index;i<n;i+=stride)
    C[i] = CMUL(A[i],CONJ(B[i]));
}

extern "C" __host__
void element_wise_multiplication_conj_2d(
    const int m,
    const int n,
    const COMPLEX *restrict A,
    const COMPLEX *restrict B,
    COMPLEX *restrict C ) {
  /*
   *  This is the CPU wrapper to the function above.
   */

  element_wise_multiplication_conj_1d_gpu<<<MIN(n,512),MIN(m,512)>>>(m*n, A, B, C);
}