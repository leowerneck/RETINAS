#include "hip/hip_runtime.h"
#include "image_analysis.h"

// GPU kernel
__global__
static void initialize_image_sum_gpu(
    const int n,
    const COMPLEX *restrict ref_image_freq,
    COMPLEX *restrict image_sum_freq ) {
  /*
   *  Compute the absolute value of all elements of an array.
   *
   *  Inputs
   *  ------
   *    n : Size of the arrays.
   *    z : Complex array of size n.
   *    x : Real array of size n. Stores the result.
   *
   *  Returns
   *  -------
   *    Nothing.
   */

  const int index  = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for(int i=index;i<n;i+=stride) {
    image_sum_freq[i].x = ref_image_freq[i].x;
    image_sum_freq[i].y = ref_image_freq[i].y;
  }
}

extern "C" __host__
void initialize_image_sum( state_struct *restrict state ) {
  /*
   *  This is the CPU wrapper to the function above.
   */
  const int Nh = state->N_horizontal;
  const int Nv = state->N_vertical;
  if( state->shot_noise_method )
    add_new_image_to_sum_gpu<<<MIN(Nv,512),MIN(Nh,512)>>>(
      state->NhNv, state->ref_image_freq, state->image_sum_freq);
  else
    add_new_image_to_sum_gpu<<<MIN(Nv,512),MIN(Nh,512)>>>(
      state->NhNv, state->reciprocal_ref_image_freq, state->image_sum_freq);
  state->image_counter = 1;
}