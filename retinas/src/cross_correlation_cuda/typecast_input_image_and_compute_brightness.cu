#include "hip/hip_runtime.h"
#include "image_analysis.h"

__global__
static void typecast_and_copy_1d_gpu(
    const int n,
    const uint16_t *restrict input_array,
    REAL *restrict real_array,
    COMPLEX *restrict complex_array) {
  /*
   *  Typecast the input image from uint16 to REAL; copy into complex array.
   *
   *  Arguments
   *  ---------
   *    Inputs
   *    ------
   *      n             : Array size.
   *      input_array   : Unsigned int16 array size n.
   *
   *    Outputs
   *    -------
   *      real_array    : Copy of the input image converted to REAL.
   *      complex_array : Copy of the input image converted to COMPLEX.
   *
   *  Returns
   *  -------
   *     Nothing.
   */

  const int index  = blockIdx.x * blockDim.x + threadIdx.x;
  const int stride = blockDim.x * gridDim.x;
  for(int i=index;i<n;i+=stride) {
    const REAL z_real = (REAL)input_array[i];
    real_array[i]     = z_real;
    complex_array[i]  = MAKE_COMPLEX(z_real,0.0);
    // Use this to initialize the image sum to zero
    image_sum_freq[i] = MAKE_COMPLEX(0.0, 0.0);
  }
}

extern "C" __host__
REAL typecast_input_image_and_compute_brightness(
    const uint16_t *restrict input_array,
    state_struct *restrict state ) {
  /*
   *  Typecast the input image from uint16 to REAL and COMPLEX. Also
   *  compute the brightness, which is the sum of all pixel values in
   *  the image. This function uses cuBLAS to compute the brightness.
   *
   *  Arguments
   *  ---------
   *    Inputs
   *    ------
   *      input_array : Input image stored as a 1D array.
   *
   *    Outputs
   *    -------
   *      state : State object; results stored in state->aux_array_real,
   *              and state->new_image_time.
   *
   *  Returns
   *  -------
   *     brightness : Brightness of the image.
   */

  // Step 1: Set useful constants
  const int Nh   = state->N_horizontal;
  const int Nv   = state->N_vertical;
  const int NhNv = Nh*Nv;

  // Step 2: Copy raw image from host (CPU) to device (GPU)
  hipMemcpy(state->aux_array_int,input_array,sizeof(uint16_t)*NhNv,hipMemcpyHostToDevice);

  // Step 3: Typecast, shift, and compute the reciprocal of the input image.
  //         We also copy the image to a real array, allowing us to use a
  //         cuBLAS functino to compute the brightness in Step 4 below.
  typecast_and_copy_1d_gpu<<<MIN(Nv,512),MIN(Nh,512)>>>(NhNv,
                                                        state->aux_array_int,
                                                        state->aux_array_real,
                                                        state->new_image_time);

  // Step 4: Compute the brightness
  REAL brightness;
  CUBLASASUM(state->cublasHandle, NhNv, state->aux_array_real, 1, &brightness);

  // Step 5: All done! Return the brightness
  return brightness;
}
