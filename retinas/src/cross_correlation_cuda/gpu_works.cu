#include "hip/hip_runtime.h"
#include "retinas.h"

__global__ void test_gpu() {
  printf("success!\n");
}

extern "C" __host__
bool gpu_works() {
  info("Testing if GPU works... ");
  fflush(stdout);
  test_gpu<<<1, 1>>>();
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if(err != hipSuccess) {
    printf("failed!\n");
    printf("Error name: %s", hipGetErrorName(err));
    printf("Error msg : %s", hipGetErrorString(err));
    return false;
  }

  return true;
}
